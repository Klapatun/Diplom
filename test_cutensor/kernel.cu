﻿#include <stdlib.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hiptensor.h"

#include <unordered_map>
#include <vector>

#include <iostream>
#include <iomanip>

using namespace std;

#define HANDLE_ERROR(x) {                                                              \
  const auto err = x;                                                                  \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                                   \
  { printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); exit(-1); } \
}

void printMatrix(float* array, int row, int column) {
    int row_offset = row / 4 - 1;
    int column_offset = column / 4 - 1;

    cout << setw(12) << "row/col";

    for (int i = 0; i < column; i += column_offset) {
        cout << setw(12) << i;
        if (i + column_offset < column) {
            cout << setw(12) << "...";
        }
    }

    cout << "\n\n";

    for (int i = 0; i < row; i += row_offset) {
        cout << setw(12) << i << ")";
        for (int j = 0; j < column; j += column_offset) {
            cout << fixed << setprecision(1) << setw(12) << array[i + (j * row)];
            if (j + column_offset < column) {
                cout << setw(12) << "...";
            }
        }
        cout << "\n";
        if (i + row_offset < row) {
            cout << setw(12) << ".\n";
            cout << setw(12) << ".\n";
            cout << setw(12) << ".\n";
        }
    }
    cout << endl;
}

int main(int argc, char** argv)
{
    srand(time(0));

    // CUDA types
    hipDataType type_big = HIP_R_32F;
    hipDataType type_core = HIP_R_32F;
    hipDataType type_data_result = HIP_R_32F;
    hiptensorComputeDescriptor_t type_compute_result = HIPTENSOR_COMPUTE_DESC_32F;

    float alpha = (float)1;
    float beta = (float)0;

    std::vector<int> mode_big{ 'm','k' };
    std::vector<int> mode_core{ 'k', 'n' };
    std::vector<int> mode_result{ 'm','n' };
    int nmodeA = mode_big.size();
    int nmodeB = mode_core.size();
    int nmodeC = mode_result.size();

    std::unordered_map<int, int64_t> extent;
    extent['m'] = 512;
    extent['k'] = 128;
    extent['n'] = 64;

    // Create a vector of extents for each tensor
    std::vector<int64_t> extent_result;
    for (auto mode : mode_result)
        extent_result.push_back(extent[mode]);

    std::vector<int64_t> extent_big;
    for (auto mode : mode_big)
        extent_big.push_back(extent[mode]);

    std::vector<int64_t> extent_core;
    for (auto mode : mode_core)
        extent_core.push_back(extent[mode]);

    // Number of elements of each tensor
    size_t elements_big = 1;
    for (auto mode : mode_big)
        elements_big *= extent[mode];
    size_t elements_core = 1;
    for (auto mode : mode_core)
        elements_core *= extent[mode];
    size_t elements_result = 1;
    for (auto mode : mode_result)
        elements_result *= extent[mode];

    size_t size_big = sizeof(float) * elements_big;
    size_t size_core = sizeof(float) * elements_core;
    size_t size_result = sizeof(float) * elements_result;

    void* big_device, * code_device, * result_device;
    hipMalloc((void**)&big_device, size_big);
    hipMalloc((void**)&code_device, size_core);
    hipMalloc((void**)&result_device, size_result);

    float* big_host = (float*)malloc(sizeof(float) * elements_big);
    float* core_host = (float*)malloc(sizeof(float) * elements_core);
    float* result_host = (float*)malloc(sizeof(float) * elements_result);
    
    for (int64_t i = 0; i < elements_big; i++) {
        big_host[i] = (float)(rand() % 1000);
    }
        
    for (int64_t i = 0; i < elements_core; i++)
        core_host[i] = (float)(rand() % 1000);
    
    for (int64_t i = 0; i < elements_result; i++)
        result_host[i] = 0;
        
        
    std::cout << "\nA:\n";

    printMatrix(big_host, extent['m'], extent['k']);
    std::cout << "\n";

    std::cout << "\nB:\n";

    printMatrix(core_host, extent['k'], extent['n']);
    std::cout << "\n";

    
    std::cout << "\nC:\n";

    printMatrix(result_host, extent['m'], extent['n']);
    std::cout << "\n";
    
    hipMemcpy(result_device, result_host, size_result, hipMemcpyHostToDevice);
    hipMemcpy(big_device, big_host, size_big, hipMemcpyHostToDevice);
    hipMemcpy(code_device, core_host, size_core, hipMemcpyHostToDevice);
    
    /* ***************************** */

    hiptensorHandle_t handle;
    cutensorInit(&handle);

    hiptensorTensorDescriptor_t desc_big;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
        &desc_big,
        nmodeA,
        extent_big.data(),
        NULL,
        type_big, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t desc_core;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
        &desc_core,
        nmodeB,
        extent_core.data(),
        NULL,
        type_core, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t desc_result;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
        &desc_result,
        nmodeC,
        extent_result.data(),
        NULL,//stride
        type_data_result, HIPTENSOR_OP_IDENTITY));

    /* ***************************** */

    uint32_t alignment_requirement_big;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
        big_device,
        &desc_big,
        &alignment_requirement_big));

    uint32_t alignment_requirement_core;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
        code_device,
        &desc_core,
        &alignment_requirement_core));

    uint32_t alignment_requirement_result;
    HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
        result_device,
        &desc_result,
        &alignment_requirement_result));

    /* ***************************** */

    cutensorContractionDescriptor_t desc;
    HANDLE_ERROR(cutensorInitContractionDescriptor(&handle,
        &desc,
        &desc_big, mode_big.data(), alignment_requirement_big,
        &desc_core, mode_core.data(), alignment_requirement_core,
        &desc_result, mode_result.data(), alignment_requirement_result,
        &desc_result, mode_result.data(), alignment_requirement_result,
        type_compute_result));

    /* ***************************** */

    cutensorContractionFind_t find;
    HANDLE_ERROR(cutensorInitContractionFind(
        &handle, &find,
        CUTENSOR_ALGO_GETT));

    /* ***************************** */

    size_t worksize = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspace(&handle,
        &desc,
        &find,
        CUTENSOR_WORKSPACE_RECOMMENDED, &worksize));

    void* work = nullptr;
    if (worksize > 0) {
        if (hipSuccess != hipMalloc(&work, worksize)) {
            work = nullptr;
            worksize = 0;
        }
    }

    /* ***************************** */

    hiptensorContractionPlan_t plan;
    HANDLE_ERROR(cutensorInitContractionPlan(&handle,
        &plan,
        &desc,
        &find,
        worksize));

    /* ***************************** */

    hiptensorStatus_t err;

    err = hiptensorContraction(&handle,
        &plan,
        (void*)&alpha, big_device,
        code_device,
        (void*)&beta, result_device,
        result_device,
        work, worksize, 0 /* stream */);
    hipDeviceSynchronize();

    std::cout << "\n" << work << "\n";

    if (err != HIPTENSOR_STATUS_SUCCESS)
    {
        printf("ERROR: %s\n", hiptensorGetErrorString(err));
    }

    hipMemcpy(result_host, result_device, size_result, hipMemcpyDeviceToHost);

    std::cout << "\n";

    printMatrix(result_host, extent['m'], extent['n']);
    std::cout << "\n";

    /* ***************************** */

    if (big_host) free(big_host);
    if (core_host) free(core_host);
    if (result_host) free(result_host);
    if (big_device) hipFree(big_device);
    if (code_device) hipFree(code_device);
    if (result_device) hipFree(result_device);
    if (work) hipFree(work);

    printf("End\n");

    return 0;
}